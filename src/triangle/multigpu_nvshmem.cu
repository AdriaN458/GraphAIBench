#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "graph_gpu.h"
#include "graph_partition.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_vertex_nvshmem.cuh"
#include <thread>

void TCSolver(Graph &g, uint64_t &total, int n_gpus, int chunk_size) {
  int ndevices = 0;
  CUDA_SAFE_CALL(hipGetDeviceCount(&ndevices));
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";

  if (ndevices < n_gpus) {
    std::cout << "Only " << ndevices << " GPUs available\n";
  } else ndevices = n_gpus;
  PartitionedGraph pg(&g, ndevices);
  pg.edgecut_partition1D();
  auto num_subgraphs = pg.get_num_subgraphs();
  int subgraph_size = (nv-1) / num_subgraphs + 1;
 
  eidType max_subg_ne = 0;
  for (int i = 0; i < ndevices; i++) {
    auto subg_ne = pg.get_subgraph(i)->E();
    if (subg_ne > max_subg_ne) 
      max_subg_ne = subg_ne;
  }
  Timer t;
  t.Start();
  GraphGPU d_graph(nv, max_subg_ne, 0, 0, 0, 0, ndevices, 1);
  d_graph.allocate_nvshmem(nv, max_subg_ne, md);
  for (int i = 0; i < ndevices; i++)
    d_graph.init_nvshmem(*pg.get_subgraph(i), i);
  t.Stop();
  std::cout << "Total GPU copy time (graph+edgelist) = " << t.Seconds() <<  " sec\n";
  int npes = nvshmem_n_pes();
  int mype = nvshmem_my_pe();
 
  size_t nthreads = BLOCK_SIZE;
  std::vector<AccType> h_counts(ndevices, 0);
  size_t nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_vertex_nvshmem, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(6*max_blocks, nblocks); 
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  std::vector<AccType *> d_count(ndevices);
  for (int i = 0; i < ndevices; i++) {
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMalloc(&d_count[i], sizeof(AccType)));
  }
  std::vector<std::thread> threads;
  std::vector<Timer> subt(ndevices);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
 
  t.Start();
  for (int i = 0; i < ndevices; i++) {
    threads.push_back(std::thread([&,i]() {
    hipSetDevice(i);
    subt[i].Start();
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(d_count[i], &h_counts[i], sizeof(AccType), hipMemcpyHostToDevice));
    vidType begin = i * subgraph_size;
    vidType end = (i+1) * subgraph_size;
    warp_vertex_nvshmem<<<nblocks, nthreads>>>(begin, end, d_graph, mype, npes, d_count[i]);
    CUDA_SAFE_CALL(hipMemcpy(&h_counts[i], d_count[i], sizeof(AccType), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    subt[i].Stop();
    }));
  }
  for (auto &thread: threads) thread.join();
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  for (int i = 0; i < ndevices; i++) total += h_counts[i];
  t.Stop();
  for (int i = 0; i < ndevices; i++)
    std::cout << "runtime[gpu" << i << "] = " << subt[i].Seconds() <<  " sec\n";
  std::cout << "runtime = " << t.Seconds() <<  " sec\n";
}

