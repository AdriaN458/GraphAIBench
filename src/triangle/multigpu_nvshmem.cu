#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "graph_gpu.h"
#include "graph_partition.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_vertex_nvshmem.cuh"
#include <thread>

long long unsigned parse_nvshmem_symmetric_size(char *value) {
  long long unsigned units, size;
  assert(value != NULL);
  if (strchr(value, 'G') != NULL) {
    units=1e9;
  } else if (strchr(value, 'M') != NULL) {
    units=1e6;
  } else if (strchr(value, 'K') != NULL) {
    units=1e3;
  } else {
    units=1;
  }
  assert(atof(value) >= 0);
  size = (long long unsigned) atof(value) * units;
  return size;
}

void TCSolver(Graph &g, uint64_t &total, int n_gpus, int chunk_size) {
  int ndevices = 0;
  CUDA_SAFE_CALL(hipGetDeviceCount(&ndevices));
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";

  if (ndevices < n_gpus) {
    std::cout << "Only " << ndevices << " GPUs available\n";
  } else ndevices = n_gpus;
  PartitionedGraph pg(&g, ndevices);
  pg.edgecut_partition1D();
  auto num_subgraphs = pg.get_num_subgraphs();
  int subgraph_size = (nv-1) / num_subgraphs + 1;

  nvshmem_init();
  int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
  std::cout << "mype_node = " << mype_node << "\n";
  hipSetDevice(mype_node);
  hipStream_t stream;
  hipStreamCreate(&stream);

  nvshmemx_init_attr_t attr;
  long long unsigned required_symmetric_heap_size = (nv+1) * sizeof(eidType) + ne * sizeof(vidType) + md * sizeof(vidType);
  printf("Setting environment variable NVSHMEM_SYMMETRIC_SIZE = %llu\n", required_symmetric_heap_size);
  char * value = getenv("NVSHMEM_SYMMETRIC_SIZE");
  if (value) { /* env variable is set */
    long long unsigned int size_env = parse_nvshmem_symmetric_size(value);
    if (size_env < required_symmetric_heap_size) {
      fprintf(stderr, "ERROR: Required > Current NVSHMEM_SYMMETRIC_SIZE=%s\n", value);
      exit(1);
    }
  } else {
    char symmetric_heap_size_str[100];
    sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
    setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);
  }
  nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

  //int npes = nvshmem_n_pes();
  //int mype = nvshmem_my_pe();
  //std::cout << "npes = " << npes << ", mype = " << mype << "\n";
  //nvshmem_barrier_all();

  eidType max_subg_ne = 0;
  for (int i = 0; i < ndevices; i++) {
    auto subg_ne = pg.get_subgraph(i)->E();
    if (subg_ne > max_subg_ne) 
      max_subg_ne = subg_ne;
  }
  Timer t;
  t.Start();
  GraphGPU d_graph(nv, max_subg_ne, 0, 0, 0, 0, ndevices, 1);
  d_graph.allocate_nvshmem(nv, max_subg_ne, md);
  for (int i = 0; i < ndevices; i++)
    d_graph.init_nvshmem(*pg.get_subgraph(i), i);
  t.Stop();
  std::cout << "Total GPU copy time (graph+edgelist) = " << t.Seconds() <<  " sec\n";
 
  size_t nthreads = BLOCK_SIZE;
  std::vector<AccType> h_counts(ndevices, 0);
  size_t nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_vertex_nvshmem, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(6*max_blocks, nblocks); 
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  std::vector<AccType *> d_count(ndevices);
  for (int i = 0; i < ndevices; i++) {
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMalloc(&d_count[i], sizeof(AccType)));
  }
  std::vector<std::thread> threads;
  std::vector<Timer> subt(ndevices);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
 
  t.Start();
  for (int i = 0; i < ndevices; i++) {
    threads.push_back(std::thread([&,i]() {
    hipSetDevice(i);
    subt[i].Start();
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(d_count[i], &h_counts[i], sizeof(AccType), hipMemcpyHostToDevice));
    vidType begin = i * subgraph_size;
    vidType end = (i+1) * subgraph_size;
    warp_vertex_nvshmem<<<nblocks, nthreads>>>(begin, end, d_graph, mype_node, ndevices, d_count[i]);
    CUDA_SAFE_CALL(hipMemcpy(&h_counts[i], d_count[i], sizeof(AccType), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    subt[i].Stop();
    }));
  }
  for (auto &thread: threads) thread.join();
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  for (int i = 0; i < ndevices; i++) total += h_counts[i];
  t.Stop();
  for (int i = 0; i < ndevices; i++)
    std::cout << "runtime[gpu" << i << "] = " << subt[i].Seconds() <<  " sec\n";
  std::cout << "runtime = " << t.Seconds() <<  " sec\n";
}

