#include "hip/hip_runtime.h"
// Copyright 2020 MIT
// Author: Xuhao Chen <cxh@mit.edu>
#include "graph_gpu.h"
#include "worklist.cuh"
#include "cuda_launch_config.hpp"

__global__ void bfs_step(GraphGPU g, vidType *dists, 
                         Worklist2 in_queue, 
                         Worklist2 out_queue) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int src;
	if (in_queue.pop_id(tid, src)) {
		int row_begin = g.edge_begin(src);
		int row_end = g.edge_end(src);
		for (int offset = row_begin; offset < row_end; ++ offset) {
			int dst = g.getEdgeDst(offset);
			if ((dists[dst] == MYINFINITY) && 
          (atomicCAS(&dists[dst], MYINFINITY, dists[src]+1) == MYINFINITY)) {
				assert(out_queue.push(dst));
			}
		}
	}
}

__global__ void insert(int source, Worklist2 queue) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id == 0) queue.push(source);
	return;
}

void BFSSolver(Graph &g, int source, vidType *h_dists) {
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";

  GraphGPU gg(g);
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (nv-1)/nthreads+1;
  if (nblocks > 65536) nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(bfs_step, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  //size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  //nblocks = std::min(max_blocks, nblocks);
  std::cout << "CUDA BFS (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  vidType zero = 0;
  vidType * d_dists;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_dists, nv * sizeof(vidType)));
  CUDA_SAFE_CALL(hipMemcpy(d_dists, h_dists, nv * sizeof(vidType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(&d_dists[source], &zero, sizeof(zero), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  Worklist2 queue1(nv), queue2(nv);
  Worklist2 *in_frontier = &queue1, *out_frontier = &queue2;
  std::cout << "CUDA BFS (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  Timer t;
  t.Start();
  int iter = 0;
  int nitems = 1;
  insert<<<1, nthreads>>>(source, *in_frontier);
  nitems = in_frontier->nitems();
  do {
    ++ iter;
    nblocks = (nitems - 1) / nthreads + 1;
    printf("iteration %d: frontier_size = %d\n", iter, nitems);
    bfs_step<<<nblocks, nthreads>>>(gg, d_dists, *in_frontier, *out_frontier);
    nitems = out_frontier->nitems();
    Worklist2 *tmp = in_frontier;
    in_frontier = out_frontier;
    out_frontier = tmp;
    out_frontier->reset();
  } while (nitems > 0);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "iterations = " << iter << ".\n";
  std::cout << "runtime [gpu_base] = " << t.Seconds() << " sec\n";
  std::cout << "throughput = " << double(ne) / t.Seconds() / 1e9 << " billion Traversed Edges Per Second (TEPS)\n";
 
  CUDA_SAFE_CALL(hipMemcpy(h_dists, d_dists, nv * sizeof(vidType), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(d_dists));
  return;
}

