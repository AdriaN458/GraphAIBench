#include "hip/hip_runtime.h"
// Copyright 2020 MIT
// Authors: Xuhao Chen <cxh@mit.edu>
#include "graph_gpu.h"
#include "utils.cuh"
#include "worklist.cuh"
#include "cuda_launch_config.hpp"
#include <hipcub/hipcub.hpp>

typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;

__global__ void insert(int source, Worklist2 queue) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id == 0) queue.push(source);
}

__device__ __forceinline__ void process_edge(GraphGPU g, int src, int edge, 
                                             elabel_t *dist, Worklist2 &outwl) {
  int dst = g.getEdgeDst(edge);
  elabel_t new_dist = dist[src] + g.getEdgeData(edge);
  if (new_dist < dist[dst]) {
    atomicMin(&dist[dst], new_dist);
    outwl.push(dst);
  }
}

__device__ void expandByCta(GraphGPU g, elabel_t *dist, Worklist2 &inwl, Worklist2 &outwl) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int vertex;
  __shared__ int owner;
  __shared__ int sh_vertex;
  owner = -1;
  int size = 0;
  if(inwl.pop_id(id, vertex)) {
    size = g.get_degree(vertex);
  }
  while(true) {
    if(size > BLOCK_SIZE)
      owner = threadIdx.x;
    __syncthreads();
    if(owner == -1)
      break;
    __syncthreads();
    if(owner == threadIdx.x) {
      sh_vertex = vertex;
      inwl.d_queue[id] = -1;
      owner = -1;
      size = 0;
    }
    __syncthreads();
    int row_begin = g.edge_begin(sh_vertex);
    int row_end = g.edge_end(sh_vertex);
    int neighbor_size = row_end - row_begin;
    int num = ((neighbor_size + blockDim.x - 1) / blockDim.x) * blockDim.x;
    for(int i = threadIdx.x; i < num; i += blockDim.x) {
      int dst = 0;
      int ncnt = 0;
      if(i < neighbor_size) {
        int offset = row_begin + i;
        dst = g.getEdgeDst(offset);
        elabel_t new_dist = dist[sh_vertex] + g.getEdgeData(offset);
        if (new_dist < dist[dst]) {
          atomicMin(&dist[dst], new_dist);
          ncnt = 1;
        }
      }
      outwl.push_1item<BlockScan>(ncnt, dst, BLOCK_SIZE);
    }
  }
}

__device__ __forceinline__ void expandByWarp(GraphGPU g, elabel_t *dist, Worklist2 &inwl, Worklist2 &outwl) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int warp_id = threadIdx.x >> LOG_WARP_SIZE;
  unsigned lane_id = LaneId();
  __shared__ int owner[NUM_WARPS];
  __shared__ int sh_vertex[NUM_WARPS];
  owner[warp_id] = -1;
  int size = 0;
  int vertex;
  if(inwl.pop_id(id, vertex)) {
    if (vertex != -1)
      size = g.get_degree(vertex);
  }
  while(__any_sync(0xFFFFFFFF, size) >= WARP_SIZE) {
    if(size >= WARP_SIZE)
      owner[warp_id] = lane_id;
    if(owner[warp_id] == lane_id) {
      sh_vertex[warp_id] = vertex;
      inwl.d_queue[id] = -1;
      owner[warp_id] = -1;
      size = 0;
    }
    int winner = sh_vertex[warp_id];
    int row_begin = g.edge_begin(winner);
    int row_end = g.edge_end(winner);
    int neighbor_size = row_end - row_begin;
    int num = ((neighbor_size + WARP_SIZE - 1) / WARP_SIZE) * WARP_SIZE;
    for(int i = lane_id; i < num; i+= WARP_SIZE) {
      int edge = row_begin + i;
      if(i < neighbor_size) {
        process_edge(g, winner, edge, dist, outwl);
      }
    }
  }
}

__global__ void bellman_ford(GraphGPU g, elabel_t *dist, Worklist2 inwl, Worklist2 outwl) {
  //expandByCta(g, dist, inwl, outwl);
  expandByWarp(g, dist, inwl, outwl);
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  int vertex;
  const int SCRATCHSIZE = BLOCK_SIZE;
  __shared__ BlockScan::TempStorage temp_storage;
  __shared__ int gather_offsets[SCRATCHSIZE];
  __shared__ int src[SCRATCHSIZE];
  gather_offsets[threadIdx.x] = 0;
  int neighborsize = 0;
  int neighboroffset = 0;
  int scratch_offset = 0;
  int total_edges = 0;
  if(inwl.pop_id(id, vertex)) {	  
    if(vertex != -1) {
      neighboroffset = g.edge_begin(vertex);
      neighborsize = g.get_degree(vertex);
    }
  }
  BlockScan(temp_storage).ExclusiveSum(neighborsize, scratch_offset, total_edges);
  int done = 0;
  int neighborsdone = 0;
  while(total_edges > 0) {
    __syncthreads();
    int i;
    for(i = 0; neighborsdone + i < neighborsize && (scratch_offset + i - done) < SCRATCHSIZE; i++) {
      gather_offsets[scratch_offset + i - done] = neighboroffset + neighborsdone + i;
      src[scratch_offset + i - done] = vertex;
    }
    neighborsdone += i;
    scratch_offset += i;
    __syncthreads();
    int edge = gather_offsets[threadIdx.x];
    if(threadIdx.x < total_edges) {
      process_edge(g, src[threadIdx.x], edge, dist, outwl);
    }
    total_edges -= BLOCK_SIZE;
    done += BLOCK_SIZE;
  }
}

void SSSPSolver(Graph &g, int source, elabel_t *h_dist, int delta) {
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";

  GraphGPU gg(g);
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (nv-1)/nthreads+1;
  assert(nblocks < 65536);
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(bellman_ford, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  //size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  //nblocks = std::min(max_blocks, nblocks);
  std::cout << "CUDA SSSP Bellman-Ford TWC (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  elabel_t zero = 0;
  elabel_t * d_dist;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_dist, nv * sizeof(elabel_t)));
  CUDA_SAFE_CALL(hipMemcpy(d_dist, h_dist, nv * sizeof(elabel_t), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(&d_dist[source], &zero, sizeof(zero), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  Worklist2 wl1(ne), wl2(ne);
  Worklist2 *inwl = &wl1, *outwl = &wl2;

  Timer t;
  t.Start();
  int iter = 0;
  int nitems = 1;
  insert<<<1, 1>>>(source, *inwl);
  nitems = inwl->nitems();
  while(nitems > 0) {
    ++ iter;
    nblocks = (nitems + BLOCK_SIZE - 1) / BLOCK_SIZE; 
    printf("iteration %d: frontier_size = %d\n", iter, nitems);
    bellman_ford<<<nblocks, BLOCK_SIZE>>>(gg, d_dist, *inwl, *outwl);
    nitems = outwl->nitems();
    Worklist2 *tmp = inwl;
    inwl = outwl;
    outwl = tmp;
    outwl->reset();
  };
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "iterations = " << iter << ".\n";
  std::cout << "runtime [sssp_gpu_twc] = " << t.Seconds() << " sec\n";
  std::cout << "throughput = " << double(ne) / t.Seconds() / 1e9 << " billion Traversed Edges Per Second (TEPS)\n";
 
  CUDA_SAFE_CALL(hipMemcpy(h_dist, d_dist, nv * sizeof(elabel_t), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipFree(d_dist));
  return;
}

void BFSSolver(Graph &g, int source, vidType *dist) {}
