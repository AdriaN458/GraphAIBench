#include "hip/hip_runtime.h"
// Copyright (c) 2022 MIT
// Author: Xuhao Chen
#include "graph_gpu.h"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "triangle_cta_compressed.cuh"

void triangle_count(Graph &g, uint64_t &total) {}

void triangle_count_compressed(Graph &g, uint64_t &total) {
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";

  GraphGPU gg(g);
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (nv-1)/nthreads+1;
  if (nblocks > 65536) nblocks = 65536;
  int max_blocks_per_SM = maximum_residency(cta_vertex_compressed, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(max_blocks, nblocks);
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  // allocate buffer for decompressed adjacency lists
  size_t per_block_buffer_size = 2 * size_t(md) * sizeof(vidType);
  size_t buffer_size = nblocks * per_block_buffer_size;
  std::cout << "buffer size: " << float(buffer_size)/float(1024*1024) << " MB\n";
  vidType *buffer;
  CUDA_SAFE_CALL(hipMalloc((void **)&buffer, buffer_size));

  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  Timer t;
  t.Start();
  cta_vertex_compressed<<<nblocks, nthreads>>>(gg, buffer, md, d_total);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime [tc_gpu_compressed] = " << t.Seconds() << " sec\n";
  std::cout << "throughput = " << double(ne) / t.Seconds() / 1e9 << " billion Traversed Edges Per Second (TEPS)\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
}

