// Copyright (c) 2022 MIT
// Author: Xuhao Chen
#include "graph_gpu.h"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
//#define TC_BS_CTA_VERTEX_COMPRESSED
#define TC_HINDEX_WARP_VERTEX_COMPRESSED
#ifdef TC_BS_CTA_VERTEX_COMPRESSED
#include "triangle_cta_compressed.cuh"
#else
#ifdef TC_HINDEX_WARP_VERTEX_COMPRESSED
#include "triangle_hindex_warp_vertex_compressed.cuh"
#endif
#endif

void triangle_count(Graph &g, uint64_t &total) {}

void triangle_count_compressed(Graph &g, uint64_t &total) {
  size_t memsize = print_device_info(0);
  GraphGPU gg(g);

  // kernel launch configuration
  size_t nthreads = BLOCK_SIZE, nblocks = (g.V()-1)/nthreads+1;
  if (nblocks > 65536) nblocks = 65536;
#ifdef TC_BS_CTA_VERTEX_COMPRESSED
  //refine_kernel_config(nthreads, nblocks, cta_vertex_compressed);
#else
  refine_kernel_config(nthreads, nblocks, hindex_warp_vertex_compressed);
#endif
  std::cout << "CUDA kernel (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  std::cout << "Allocating buffer for decompressed adjacency lists\n";
  vidType *buffer;
  size_t num_per_block = WARPS_PER_BLOCK;
#ifdef TC_BS_CTA_VERTEX_COMPRESSED
  num_per_block = 1;
#endif
  allocate_gpu_buffer(3 * size_t(g.get_max_degree()) * num_per_block * nblocks, buffer);

#ifdef TC_HINDEX_WARP_VERTEX_COMPRESSED
  std::cout << "Allocating buckets for the hash map\n";
  vidType *bins;
  allocate_gpu_buffer(NUM_BUCKETS * BUCKET_SIZE * WARPS_PER_BLOCK * nblocks, bins);
  auto bins_mem = NUM_BUCKETS * BUCKET_SIZE * WARPS_PER_BLOCK * nblocks * sizeof(vidType);
  CUDA_SAFE_CALL(hipMemset(bins, 0, bins_mem));
#endif

  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  Timer t;
  t.Start();
#ifdef TC_BS_CTA_VERTEX_COMPRESSED
  cta_vertex_compressed<<<nblocks, nthreads>>>(gg, buffer, g.get_max_degree(), d_total);
#else
#ifdef TC_HINDEX_WARP_VERTEX_COMPRESSED
  std::cout << "Use HINDEX\n";
  hindex_warp_vertex_compressed<<<nblocks, nthreads>>>(gg, bins, buffer, g.get_max_degree(), d_total);
  //hindex_warp_vertex<<<nblocks, nthreads>>>(gg, bins, d_total);
#endif
#endif
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime = " << t.Seconds() << " sec\n";
  std::cout << "throughput = " << double(g.E()) / t.Seconds() / 1e9 << " billion Traversed Edges Per Second (TEPS)\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
  CUDA_SAFE_CALL(hipFree(buffer));
#ifdef TC_HINDEX_WARP_VERTEX_COMPRESSED
  CUDA_SAFE_CALL(hipFree(bins));
#endif
}

