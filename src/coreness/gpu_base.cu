#include <hipcub/hipcub.hpp>
#include "timer.h"
#include "graph_gpu.h"
#include "operations.cuh"
#include "hip/hip_runtime_api.h"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;

void KCoreSolver(Graph &g, std::vector<int> &coreness, vidType &largest_core, int, int) {
}
